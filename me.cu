#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "dsp.h"
#include "me.cuh"

/* Motion estimation for 8x8 block */
__global__ static void me_block_8x8(struct c63_common *cm,uint8_t *orig, uint8_t *ref, int color_component)
{
  //struct macroblock *mb =&cm->curframe->mbs[color_component][mb_y*cm->padw[color_component]/8+mb_x];
  int range = cm->me_search_range;

    __shared__ int sIdx[8][8];
   //thread index
  int x_index = threadIdx.x;
  int y_index = threadIdx.y;
 
  //block index
  int block_y = blockIdx.x;
  int block_x = blockIdx.y;


  struct macroblock *mb  = &cm->curframe->mbs[color_component][block_y*cm->padw[color_component]/8+block_x];

  /* Quarter resolution for chroma channels. */
  if (color_component > 0) { range /= 2; }

  int left = block_x * 8 - range;
  int top = block_y * 8 - range;
  int right = block_x * 8 + range;
  int bottom = block_y * 8 + range;

  int w = cm->padw[color_component];
  int h = cm->padh[color_component];

  /* Make sure we are within bounds of reference frame. TODO: Support partial
     frame bounds. */
  if (left < 0) { left = 0; }
  if (top < 0) { top = 0; }
  if (right > (w - 8)) { right = w - 8; }
  if (bottom > (h - 8)) { bottom = h - 8; }

  int x, y;

  uint8_t *bl1, *bl2;
  int sad;
  int mx = block_x * 8;
  int my = block_y * 8;

  int best_sad = INT_MAX;

  for (y = top; y < bottom; ++y)
  {
    for (x = left; x < right; ++x)
    {
      sad=0;
      bl1 = orig + my*w+mx;
      bl2 = ref + y*w+x;
      //sad_block_8x8(orig + my*w+mx, ref + y*w+x, w, &sad);

      //here, sad_block_8x8 will be replaced, so that each thread can calculate their own sum absolute differences

      /* printf("(%4d,%4d) - %d\n", x, y, sad); */
      __syncthreads();
       sIdx[x_index][y_index] = abs(bl2[x_index * w + y_index] - bl1[x_index * w + y_index]);
      __syncthreads();


      //y axis
      if(y_index < 4){
        sIdx[x_index][y_index] += sIdx[x_index][7-y_index];
      }
      __syncthreads();
      if (y_index < 2) {
        sIdx[x_index][y_index] += sIdx[x_index][3-y_index];
      }
      __syncthreads();
      if (y_index < 1){
        sIdx[x_index][y_index] += sIdx[x_index][1];
      }
      __syncthreads();

      // reduce along x axis
      if(x_index < 4  && y_index == 0){
        sIdx[x_index][0] += sIdx[7-x_index][0];
      }
      __syncthreads();
      if (x_index < 2  && y_index== 0) {
        sIdx[x_index][0] += sIdx[3-x_index][0];
      }
      __syncthreads();
      if (x_index< 1 && y_index == 0){
        sIdx[x_index][y_index] += sIdx[1][y_index];
      }
      //sum
       if (x_index ==0 && y_index == 0) {
        
        sad = sIdx[0][0];

        if (sad < best_sad)
          {
          mb->mv_x = x - mx;
          mb->mv_y = y - my;
          best_sad = sad;
        }
       
     }
    }
  }
  /* Here, there should be a threshold on SAD that checks if the motion vector
     is cheaper than intraprediction. We always assume MV to be beneficial */

  /* printf("Using motion vector (%d, %d) with SAD %d\n", mb->mv_x, mb->mv_y,
     best_sad); */

  mb->use_mv = 1;
}

__global__ void c63_motion_estimate(struct c63_common *cm)
{
  /* Compare this frame with previous reconstructed frame */
  //int mb_x, mb_y;

  dim3 threads(8,8);

  /* Luma */
 
  if (threadIdx.x == 0){
    
    dim3 y_dim (cm->mb_rows, cm->mb_cols);
    me_block_8x8 <<<y_dim, threads>>>(cm, cm->curframe->orig->Y,cm->refframe->recons->Y, Y_COMPONENT);
    return; 
  }

    /* Chroma */

  if (threadIdx.x == 1){
    
    dim3 UV_dim(cm->mb_rows / 2, cm->mb_cols / 2);

    me_block_8x8<<<UV_dim, threads>>> (cm, cm->curframe->orig->U,cm->refframe->recons->U, U_COMPONENT);
    return; 
  }
  // V
  if (threadIdx.x == 2){
    
    dim3 UV_dim(cm->mb_rows / 2, cm->mb_cols / 2);

    me_block_8x8<<<UV_dim, threads>>> (cm, cm->curframe->orig->V,
      cm->refframe->recons->V, V_COMPONENT);


    return;  
  }


}

/* Motion compensation for 8x8 block */
__global__ static void mc_block_8x8(struct c63_common *cm, uint8_t *predicted, uint8_t *ref, int color_component)
{
    
  int mb_x = blockIdx.y;
  int mb_y = blockIdx.x;
  struct macroblock *mb =&cm->curframe->mbs[color_component][mb_y*cm->padw[color_component]/8+mb_x];

  if (!mb->use_mv) { return; }

  int left = mb_x * 8;
  int top = mb_y * 8;
  int right = left + 8;
  int bottom = top + 8;

  int w = cm->padw[color_component];

  /* Copy block from ref mandated by MV */
  int x, y;

  for (y = top; y < bottom; ++y)
  {
    for (x = left; x < right; ++x)
    {
      predicted[y*w+x] = ref[(y + mb->mv_y) * w + (x + mb->mv_x)];
    }
  }
}

__global__ void c63_motion_compensate(struct c63_common *cm)
{
 /* Compare this frame with previous reconstructed frame */
  //int mb_x, mb_y;

  dim3 threads(8,8);

  /* Luma */
 
  if (threadIdx.x == 0){
    
    dim3 y_dim (cm->mb_rows, cm->mb_cols);
    mc_block_8x8 <<<y_dim, threads>>>(cm, cm->curframe->orig->Y,cm->refframe->recons->Y, Y_COMPONENT);
    return; 
  }

    /* Chroma */

  if (threadIdx.x == 1){
    
    dim3 UV_dim(cm->mb_rows / 2, cm->mb_cols / 2);

    mc_block_8x8<<<UV_dim, threads>>> (cm, cm->curframe->orig->U,cm->refframe->recons->U, U_COMPONENT);
    return; 
  }
  // V
  if (threadIdx.x == 2){
    
    dim3 UV_dim(cm->mb_rows / 2, cm->mb_cols / 2);

    mc_block_8x8<<<UV_dim, threads>>> (cm, cm->curframe->orig->V,
      cm->refframe->recons->V, V_COMPONENT);


    return;  
  }

}
